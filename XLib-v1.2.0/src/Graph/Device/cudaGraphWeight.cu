#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------------
Copyright © 2016 by Nicola Bombieri

XLib is provided under the terms of The MIT License (MIT):

Permission is hereby granted, free of charge, to any person obtaining a copy of
this software and associated documentation files (the "Software"), to deal in
the Software without restriction, including without limitation the rights to
use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
the Software, and to permit persons to whom the Software is furnished to do so,
subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
------------------------------------------------------------------------------*/
/**
 * @author Federico Busato
 * Univerity of Verona, Dept. of Computer Science
 * federico.busato@univr.it
 */
#include "../include/cudaGraphNamespace.cuh"
#include "../include/cudaGraphWeight.cuh"

namespace cuda_graph {

cudaGraphWeight::cudaGraphWeight(GraphWeight& _graph,
                                 const bool _inverse_graph,
                                 const int _degree_options) :
                                 graph(_graph),
                                 inverse_graph(_inverse_graph),
                                 degree_options(_degree_options) {

    hipMalloc(&devOutNodes, (graph.V + 1) * sizeof(edge_t));
    hipMalloc(&devOutEdges, graph.E * sizeof(edgew_t));

    if (inverse_graph) {
        hipMalloc(&devInNodes, (graph.V + 1) * sizeof(edge_t));
        hipMalloc(&devInEdges, graph.E * sizeof(node_t));
    }

    if ((degree_options & IN_DEGREE) && (degree_options & OUT_DEGREE))
        hipMalloc(&devInOutDegrees, graph.V * sizeof(int2));
    else if (degree_options & IN_DEGREE)
        hipMalloc(&devInDegrees, graph.V * sizeof(degree_t));
    else if (degree_options & OUT_DEGREE)
        hipMalloc(&devOutDegrees, graph.V * sizeof(degree_t));

    __CUDA_ERROR("Graph Allocation");
}

cudaGraphWeight::~cudaGraphWeight() {
    hipFree(devOutNodes);
    hipFree(devOutEdges);

    if (inverse_graph) {
        hipFree(devInNodes);
        hipFree(devInEdges);
    }
    if ((degree_options & IN_DEGREE) && (degree_options & OUT_DEGREE))
        hipFree(devInOutDegrees);
    else if (degree_options & IN_DEGREE)
        hipFree(devInDegrees);
    else if (degree_options & OUT_DEGREE)
        hipFree(devOutDegrees);
}

void cudaGraphWeight::copyToDevice() {
    hipMemcpy(devOutNodes, graph.OutNodes, (graph.V + 1) * sizeof(edge_t), hipMemcpyHostToDevice);

    int2* WeightOutEdge = new int2[graph.E];
    for (int i = 0; i < graph.E; i++)
        WeightOutEdge[i] = make_int2(graph.OutEdges[i], graph.Weights[i]);

    hipMemcpy(devOutEdges, WeightOutEdge, graph.E * sizeof(edgew_t), hipMemcpyHostToDevice);
    delete[] WeightOutEdge;

    if (inverse_graph) {
        hipMemcpy(devInNodes, graph.InNodes, (graph.V + 1) * sizeof(edge_t), hipMemcpyHostToDevice);
        hipMemcpy(devInEdges, graph.InEdges, graph.E * sizeof(node_t), hipMemcpyHostToDevice);
    }

    if ((degree_options & IN_DEGREE) && (degree_options & OUT_DEGREE)) {
        int2* tmpInOutDegrees = new int2[graph.V];
        for (int i = 0; i < graph.V; i++)
            tmpInOutDegrees[i] = make_int2(graph.InDegrees[i], graph.OutDegrees[i]);

        hipMemcpy(devInOutDegrees, tmpInOutDegrees, graph.V * sizeof(int2), hipMemcpyHostToDevice);
        delete[] tmpInOutDegrees;
    }
    else if (degree_options & IN_DEGREE)
        hipMemcpy(devInDegrees, graph.InDegrees, graph.V * sizeof(degree_t), hipMemcpyHostToDevice);
    else if (degree_options & OUT_DEGREE)
        hipMemcpy(devOutDegrees, graph.OutDegrees, graph.V * sizeof(degree_t), hipMemcpyHostToDevice);

    __CUDA_ERROR("Graph Copy To Device");
}

} //@cuda_graph
